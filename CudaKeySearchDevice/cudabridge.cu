#include "cudabridge.h"

#include "puzzle71_kernel.h"

void callKeyFinderKernel(int blocks, int threads, int points, bool /*useDouble*/, int compression)
{
    dim3 grid(static_cast<unsigned int>(blocks), 1, 1);
    dim3 block(static_cast<unsigned int>(threads), 1, 1);

    hipError_t launch_status = puzzle71::kernel::LaunchFusedKernel(grid, block, points, compression);
    if (launch_status != hipSuccess) {
        throw cuda::CudaException(launch_status);
    }

    hipError_t sync_status = hipDeviceSynchronize();
    if (sync_status != hipSuccess) {
        throw cuda::CudaException(sync_status);
    }
}

void waitForKernel()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw cuda::CudaException(err);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        throw cuda::CudaException(err);
    }
}
